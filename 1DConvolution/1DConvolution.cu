#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void convolution_1d_kernel(const float* input, const float* kernel, float* output,
                                      int input_size, int kernel_size) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < input_size - kernel_size+1){
        float Cvalue = 0.0;
        for(int i=0;i<kernel_size;++i){
            Cvalue += input[idx+i] * kernel[i];
        }
        output[idx]=Cvalue;
    }

}

// input, kernel, output are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* input, const float* kernel, float* output, int input_size, int kernel_size) {
    int output_size = input_size - kernel_size + 1;
    int threadsPerBlock = 256;
    int blocksPerGrid = (output_size + threadsPerBlock - 1) / threadsPerBlock;

    convolution_1d_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, kernel, output, input_size, kernel_size);
    hipDeviceSynchronize();
}